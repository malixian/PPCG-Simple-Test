#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <iostream>

void print_matrix(float* mat, int row, int col) {
    for (int i = 0; i < row * col; i++) {
        printf("%f\t", mat[i]);
        if ((i+1) % col == 0) {
            printf("\n");
        }

    }
     printf("----------------------------\n");
}


void cpu_mm(float *mat1, float *mat2, float* result){
        int m_size  = 32*256;
        for (int r = 0; r < 1; r++) {
        for (int c = 0; c < 10; c++) {
            for (int n = 0; n < m_size; n++) {
                result[r*m_size + c] += mat1[r*m_size+n] * mat2[n*m_size+c];
            }
        }

        }
}

 int check(float *c_result, float *g_result) {
     for(int i=0; i<10; i++)
         if(c_result[i] != g_result[i]){
             std::cout<<"check failed, original is:"<<c_result[i]<<"result is:"<<g_result[i]<<std::endl;
         }

 }


int main(int argc, char *argv[]) {
    float *mat1, *mat2, *result;
    float *g_mat1, *g_mat2, *g_mat_result;
    int r_size, m_size;    // 矩阵行数，矩阵size

    hipError_t cudaStat;
    hipblasHandle_t handle;
    hipblasStatus_t stat;

    if (argc > 1) {
        r_size = atoi(argv[1]);
    } else {
        r_size = 8192;
    }
    m_size = r_size * r_size;

    // 用一位数组表示二维矩阵
    mat1 = (float*) malloc(m_size * sizeof(float));
    mat2 = (float*) malloc(m_size * sizeof(float));
    result = (float*) malloc(m_size * sizeof(float));

    // initialize
    for (int i = 0; i < m_size; i++) {
        mat1[i] = rand()/10000000;
        mat2[i] = rand()/10000000;
        result[i] = 0;
    }

    cudaStat = hipMalloc((void **)&g_mat1, sizeof(*mat1) * m_size);
    cudaStat = hipMalloc((void **)&g_mat2, sizeof(*mat2) * m_size);
    cudaStat = hipMalloc((void **)&g_mat_result, sizeof(*result) * m_size);
    printf("cudaStat %d\n", cudaStat);

    // initialize CUBLAS context
    stat = hipblasCreate(&handle);

    stat = hipblasSetMatrix(r_size, r_size, sizeof(*mat1), mat1, r_size, g_mat1, r_size);
    stat = hipblasSetMatrix(r_size, r_size, sizeof(*mat2), mat2, r_size, g_mat2, r_size);
    stat = hipblasSetMatrix(r_size, r_size, sizeof(*result), result, r_size, g_mat_result, r_size);

    float al = 1.0f;
    float bet = 0.0f;

         hipEvent_t start, stop;
     float time;
     hipEventCreate(&start);
     hipEventCreate(&stop);
     hipEventRecord( start, 0 );

    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        r_size, r_size, r_size, &al, g_mat1,
        r_size, g_mat2, r_size, &bet, g_mat_result, r_size);

        hipEventRecord( stop, 0  );
    hipEventSynchronize( start );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    std::cout<<"cost time is:"<<time<<std::endl;
    hipEventDestroy( start );
    hipEventDestroy( stop );

    stat = hipblasGetMatrix(r_size, r_size, sizeof(*result), g_mat_result, r_size, result, r_size);
    printf("cublas %d\n",stat);
    // cudaMemcpy(result, g_mat_result, sizeof(float) * m_size, cudaMemcpyDeviceToHost);

        float* c_result = (float*) malloc(m_size * sizeof(float));
        cpu_mm(mat1, mat2, c_result);
        check(c_result, result);
    if (r_size < 10) {
        printf("-----mat1----\n");
        print_matrix(mat1, r_size, r_size);
        printf("-----mat2----\n");
        print_matrix(mat2, r_size, r_size);
        printf("----mat1 * mat2---\n");
        print_matrix(result, r_size, r_size);
    }
    hipFree(g_mat1);
    hipFree(g_mat2);
    hipFree(g_mat_result);
    free(mat1);
    free(mat2);
    free(result);
}

