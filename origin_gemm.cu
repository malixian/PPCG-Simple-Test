#include "hip/hip_runtime.h"
nclude <stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include <iostream>



#define BLOCK_NUM 8   //块数量
#define THREAD_NUM 128 // 每个块中的线程数
#define R_SIZE BLOCK_NUM * THREAD_NUM
#define M_SIZE R_SIZE * R_SIZE

__global__ void mat_mul(int *mat1, int *mat2, int *result) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    // 每个线程计算一行
    const int gid = bid * THREAD_NUM + tid;
    const int row = gid / R_SIZE;
    const int col = gid % R_SIZE;
    for (int n = 0; n < R_SIZE; n++) {
        result[row*R_SIZE+col] += mat1[row*R_SIZE+n] * mat2[n*R_SIZE+col];
    }

}

int main(int argc, char *argv[]) {
    int *mat1, *mat2, *result;
    int *g_mat1, *g_mat2, *g_mat_result;

    // 用一位数组表示二维矩阵
    mat1 = (int*) malloc(M_SIZE * sizeof(int));
    mat2 = (int*) malloc(M_SIZE * sizeof(int));
    result = (int*) malloc(M_SIZE * sizeof(int));

    // initialize
    for (int i = 0; i < M_SIZE; i++) {
        mat1[i] = rand()/1000000;
        mat2[i] = rand()/1000000;
        result[i] = 0;

    }

    hipMalloc((void **)&g_mat1, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_mat2, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_mat_result, sizeof(int) * M_SIZE);

    hipMemcpy(g_mat1, mat1, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(g_mat2, mat2, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);

        hipEvent_t start, stop;
    float time = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );

    mat_mul<<<BLOCK_NUM, THREAD_NUM>>>(g_mat1, g_mat2, g_mat_result);

        hipEventRecord( stop, 0  );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    std::cout<<"cost time is:"<<time<<std::endl;
    hipEventDestroy( start );
    hipEventDestroy( stop );


    hipMemcpy(result, g_mat_result, sizeof(int) * M_SIZE, hipMemcpyDeviceToHost);
}

